#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2023 Xiaoyang Chen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software
// and associated documentation files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all copies or
// substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include "libllm/cuda/common.h"
#include "libllm/cuda/transform.h"

namespace libllm {
namespace op {
namespace cuda {

template<typename T>
__global__ void transform5DKernel(
    PackedSubtensor<const T, 5> src,
    T alpha,
    T beta,
    PackedSubtensor<T, 5> dest) {
  int d4 = blockIdx.x * blockDim.x + threadIdx.x;
  int d3 = blockIdx.y * blockDim.y + threadIdx.y;

  dim3 dz = splitIndexToDim3(blockIdx.z * blockDim.z + threadIdx.z, src.getSize());
  int d2 = dz.x;
  int d1 = dz.y;
  int d0 = dz.z;

  const Size *s = src.getSize();

  if (d0 < s[0].shape && d1 < s[1].shape && d2 < s[2].shape && d3 < s[3].shape && d4 < s[4].shape) {
    dest[d0][d1][d2][d3][d4] = alpha * src[d0][d1][d2][d3][d4] + beta;
  }
}

template<typename T>
__global__ void transform4DKernel(
    PackedSubtensor<const T, 4> src,
    T alpha,
    T beta,
    PackedSubtensor<T, 4> dest) {
  int d3 = blockIdx.x * blockDim.x + threadIdx.x;

  dim3 dz = splitIndexToDim3(blockIdx.y * blockDim.y + threadIdx.y, src.getSize());
  int d2 = dz.x;
  int d1 = dz.y;
  int d0 = dz.z;

  const Size *s = src.getSize();

  if (d0 < s[0].shape && d1 < s[1].shape && d2 < s[2].shape && d3 < s[3].shape) {
    dest[d0][d1][d2][d3] = alpha * src[d0][d1][d2][d3] + beta;
  }
}

template<typename T>
__global__ void transform3DKernel(
    PackedSubtensor<const T, 3> src,
    T alpha,
    T beta,
    PackedSubtensor<T, 3> dest) {
  int d2 = blockIdx.x * blockDim.x + threadIdx.x;
  int d1 = blockIdx.y * blockDim.y + threadIdx.y;
  int d0 = blockIdx.z * blockDim.z + threadIdx.z;

  const Size *s = src.getSize();

  if (d0 < s[0].shape && d1 < s[1].shape && d2 < s[2].shape) {
    dest[d0][d1][d2] = alpha * src[d0][d1][d2] + beta;
  }
}

template<typename T>
void transform5D(Tensor src, Tensor dest, T alpha, T beta) {
  src.throwIfInvalidShape(dest.getShape(), "transform5D");

  PackedSubtensor<const T, 5> sA(src);
  PackedSubtensor<T, 5> sC(dest);

  constexpr int blockSize = 256;
  dim3 d;
  d.z = src.getShape(0) * src.getShape(1) * src.getShape(2);
  d.y = src.getShape(3);
  d.x = (src.getShape(4) + blockSize - 1) / blockSize;

  transform5DKernel<T><<<d, blockSize>>>(sA, alpha, beta, sC);
  hipDeviceSynchronize();
  LL_CHECK_CUDA_STATUS(hipGetLastError());
}

template<typename T>
void transform4D(Tensor src, Tensor dest, T alpha, T beta) {
  src.throwIfInvalidShape(dest.getShape(), "transform4D");

  PackedSubtensor<const T, 4> sA(src);
  PackedSubtensor<T, 4> sC(dest);

  constexpr int blockSize = 256;
  dim3 d;
  d.y = src.getShape(0) * src.getShape(1) * src.getShape(2);
  d.x = (src.getShape(3) + blockSize - 1) / blockSize;

  transform4DKernel<T><<<d, blockSize>>>(sA, alpha, beta, sC);
  hipDeviceSynchronize();
  LL_CHECK_CUDA_STATUS(hipGetLastError());
}

template<typename T>
void transform3D(Tensor src, Tensor dest, T alpha, T beta) {
  src.throwIfInvalidShape(dest.getShape(), "transform3D");

  PackedSubtensor<const T, 3> sA(src);
  PackedSubtensor<T, 3> sC(dest);

  constexpr int blockSize = 256;
  dim3 d;
  d.z = src.getShape(0);
  d.y = src.getShape(1);
  d.x = (src.getShape(2) + blockSize - 1) / blockSize;

  transform3DKernel<T><<<d, blockSize>>>(sA, alpha, beta, sC);
  hipDeviceSynchronize();
  LL_CHECK_CUDA_STATUS(hipGetLastError());
}

template<typename T>
void transform2D(Tensor src, Tensor dest, T alpha, T beta) {
  src.throwIfInvalidShape(dest.getShape(), "transform2D");

  int d0 = src.getShape(0);
  int d1 = src.getShape(1);

  return transform3D(src.view({1, d0, d1}), dest.view({1, d0, d1}), alpha, beta);
}

template<typename T>
void transform1D(Tensor src, Tensor dest, T alpha, T beta) {
  src.throwIfInvalidShape(dest.getShape(), "transform2D");

  int d0 = src.getShape(0);
  return transform3D(src.view({1, 1, d0}), dest.view({1, 1, d0}), alpha, beta);
}

void transformHalf(Tensor src, Tensor dest, half alpha, half beta) {
  CHECK(src.getDType() == DType::kFloat16);
  CHECK(dest.getDType() == DType::kFloat16);

  if (src.getDim() == 5) return transform5D<half>(src, dest, alpha, beta);
  if (src.getDim() == 4) return transform4D<half>(src, dest, alpha, beta);
  if (src.getDim() == 3) return transform3D<half>(src, dest, alpha, beta);
  if (src.getDim() == 2) return transform2D<half>(src, dest, alpha, beta);
  if (src.getDim() == 1) return transform1D<half>(src, dest, alpha, beta);

  NOT_IMPL();
}

Tensor transform(const Tensor &src, float alpha, float beta) {
  CHECK(src.getDevice().getType() == Device::kCuda);

  if (src.getDType() == DType::kFloat16) {
    Tensor dest = createCudaTensorHalf(src.getShape());
    transformHalf(src, dest, half(alpha), half(beta));
    return dest;
  } else {
    NOT_IMPL();
  }
}

}  // namespace cuda
}  // namespace op
}  // namespace libllm
