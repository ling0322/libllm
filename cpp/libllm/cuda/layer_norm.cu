#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2023 Xiaoyang Chen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software
// and associated documentation files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all copies or
// substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include <hip/hip_fp16.h>

#include "libllm/cuda/binary_op.h"
#include "libllm/cuda/cast.h"
#include "libllm/cuda/common.h"
#include "libllm/cuda/layer_norm.h"
#include "libllm/cuda/reduce.h"
#include "libllm/cuda/transform.h"
#include "libllm/functional.h"

namespace libllm {
namespace op {
namespace cuda {

__global__ void layerNormKernel3D(
    PackedSubtensor<const half, 3> inputTensor,
    PackedSubtensor<const half, 2> mean,
    PackedSubtensor<const float, 2> sumDiffSquare,
    PackedSubtensor<const half, 1> weight,
    PackedSubtensor<const half, 1> bias,
    PackedSubtensor<half, 3> outputTensor,
    float eps) {
  assert(inputTensor.getShape(0) == outputTensor.getShape(0));
  assert(inputTensor.getShape(1) == outputTensor.getShape(1));
  assert(inputTensor.getShape(2) == outputTensor.getShape(2));
  assert(inputTensor.getShape(0) == mean.getShape(0));
  assert(inputTensor.getShape(1) == mean.getShape(1));
  assert(inputTensor.getShape(0) == sumDiffSquare.getShape(0));
  assert(inputTensor.getShape(1) == sumDiffSquare.getShape(1));
  assert(inputTensor.getShape(2) == weight.getShape(0));
  assert(inputTensor.getShape(2) == bias.getShape(0));

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (z < inputTensor.getShape(0) && y < inputTensor.getShape(1) && x < inputTensor.getShape(2)) {
    float meanVal = mean[z][y];
    float varVal = sumDiffSquare[z][y] / inputTensor.getShape(2);
    float sd = sqrt(varVal + eps);
    float elem = inputTensor[z][y][x];
    elem = (elem - meanVal) / sd;
    elem = elem * __half2float(weight[x]) + __half2float(bias[x]);
    outputTensor[z][y][x] = elem;
  }
}

Tensor layerNorm3D(Tensor tensor, Tensor weight, Tensor bias, float eps) {
  CHECK(tensor.getShape(-1) == weight.getShape(0) && bias.getShape(0) == weight.getShape(0));

  Tensor reduceSum = op::cuda::reduceHalfToSingle3D(tensor, MapReduceType::SUM_FP16_FP32);
  CHECK(reduceSum.getDim() == 2);
  reduceSum = op::cuda::castFloatToHalf(reduceSum);

  Tensor mean = op::cuda::transform(reduceSum, 1.0 / tensor.getShape(2), 0.0f);
  Tensor diff = op::cuda::binaryOp(tensor, mean.unsqueeze(2), BinaryOp::SUB);

  Tensor sumDiffSquare = op::cuda::reduceHalfToSingle3D(diff, MapReduceType::SUM_SQUARE_FP16_FP32);
  Tensor C = createCudaTensorHalf(tensor.getShape());

  constexpr int blockSize = 256;
  dim3 d;
  d.z = C.getShape(0);
  d.y = C.getShape(1);
  d.x = (C.getShape(2) + blockSize - 1) / blockSize;

  layerNormKernel3D<<<d, blockSize>>>(tensor, mean, sumDiffSquare, weight, bias, C, eps);
  hipDeviceSynchronize();
  LL_CHECK_CUDA_STATUS(hipGetLastError());
  return C;
}

Tensor layerNorm(Tensor tensor, Tensor weight, Tensor bias, float eps) {
  CHECK(weight.getDim() == 1 && bias.getDim() == 1);
  CHECK(tensor.getShape(-1) == weight.getShape(0));
  CHECK(tensor.getShape(-1) == bias.getShape(0));
  CHECK(weight.getDevice().getType() == Device::kCuda);

  if (tensor.getDim() == 3) return layerNorm3D(tensor, weight, bias, eps);

  NOT_IMPL();
}

}  // namespace cuda
}  // namespace op
}  // namespace libllm
