#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2023 Xiaoyang Chen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software
// and associated documentation files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in all copies or
// substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.


#include <hip/hip_fp16.h>
#include <math.h>
#include "libllm/functional.h"
#include "libllm/cuda/common.h"
#include "libllm/cuda/reduce.h"

namespace libllm {
namespace op {
namespace cuda {

template<typename T>
__global__ void softmaxKernel3D(PackedSubtensor<const T, 3> input,
                                PackedSubtensor<const float, 2> sumExp,
                                PackedSubtensor<T, 3> output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < input.getShape(2) && y < input.getShape(1) && z < input.getShape(0)) {
    float el = static_cast<float>(input[z][y][x]);
    output[z][y][x] = static_cast<T>(expf(el - logf(sumExp[z][y])));
  }
}

Tensor softmaxHalf3D(Tensor A) {
  CHECK(A.getDType() == DType::kFloat16);
  CHECK(A.getDim() == 3);

  Tensor sumExp = reduceHalfToSingle3D(A, ReduceType::SUM_EXP_FP16_FP32);
  Tensor C = createCudaTensorHalf(A.getShape());

  constexpr int blockSize = 256;
  dim3 d;
  d.z = A.getShape(0);
  d.y = A.getShape(1);
  d.x = (A.getShape(2) + blockSize - 1) / blockSize;

  softmaxKernel3D<half><<<d, blockSize>>>(A, sumExp, C);
  hipDeviceSynchronize();
  LL_CHECK_CUDA_STATUS(hipGetLastError());

  return C;
}

Tensor softmaxHalf1D(Tensor A) {
  Tensor xA = A.view({1, 1, A.getShape(0)});
  Tensor C = softmaxHalf3D(xA);

  return C.view({C.getShape(2)});
}

Tensor softmaxHalf2D(Tensor A) {
  Tensor xA = A.view({1, A.getShape(0), A.getShape(1)});
  Tensor C = softmaxHalf3D(xA);

  return C.view({C.getShape(1), C.getShape(2)});
}

Tensor softmaxHalf4D(Tensor A) {
  std::vector<int> shape = A.getShape();

  Tensor xA = A.view({-1, A.getShape(2), A.getShape(3)});
  Tensor C = softmaxHalf3D(xA);

  return C.view(shape);
}

Tensor softmaxHalf(Tensor A) {
  if (A.getDim() == 1) return softmaxHalf1D(A);
  if (A.getDim() == 2) return softmaxHalf2D(A);
  if (A.getDim() == 3) return softmaxHalf3D(A);
  if (A.getDim() == 4) return softmaxHalf4D(A);

  NOT_IMPL();
}

Tensor softmax(Tensor A) {
  if (A.getDType() == DType::kFloat16) return softmaxHalf(A);

  NOT_IMPL();
}

}  // cuda
}  // op
}  // ly
