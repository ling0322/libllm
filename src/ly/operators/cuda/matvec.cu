#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2023 Xiaoyang Chen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software
// and associated documentation files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy, modify, merge, publish,
// distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in all copies or
// substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

// Inspired by https://github.com/ankan-ban/llama_cu_awq and https://github.com/mit-han-lab/llm-awq

#include <hip/hip_fp16.h>
#include "ly/tensor.h"
#include "ly/operators/cuda/common.h"

namespace ly {
namespace op {
namespace cuda {

int divUp(int a, int b) {
    return (a - 1) / b + 1;
} 

// load with cache streaming.
__forceinline__ __device__ void load16byteCS(const void *src, void *dest) {
  uint4 d = __ldcs((const uint4 *)src);
  *((uint4 *)dest) = d;
}

template<typename T>
__forceinline__ __device__ void load16byte(const void *src, T *dest) {
  static_assert(sizeof(T) == 16, "T is not a 16 byte struct");
  uint4 d = *((const uint4 *)src);
  *((uint4 *)dest) = d;
}

__device__ __forceinline__ float wrapReduceSum(float sum) {
  sum += __shfl_down_sync(0xffffffff, sum, 16);
  sum += __shfl_down_sync(0xffffffff, sum, 8);
  sum += __shfl_down_sync(0xffffffff, sum, 4);
  sum += __shfl_down_sync(0xffffffff, sum, 2);
  sum += __shfl_down_sync(0xffffffff, sum, 1);
  return sum;
}

__global__ void mat_vec_kernel(half* y, const half *__restrict__ x, const half *__restrict__ A, int n, int d, int lda) {
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  if (row >= d)
    return;

  constexpr int Vec = 8;
  constexpr int WrapSize = 32;
  int startIdx = threadIdx.x * Vec;

  float sum = 0;
  for (int i = startIdx; i < n; i += Vec * WrapSize) {
    half packA[Vec];
    half packX[Vec];
    load16byteCS(&A[row * lda + i], &packA);
    load16byte<half[Vec]>(&x[i], &packX);

    #pragma unroll
    for (int j = 0; j < Vec; j++)
      sum += float(packA[j]) * float(packX[j]);
  }

  sum = wrapReduceSum(sum);
  if (threadIdx.x == 0)
    y[row] = (half)sum;
}

__global__ void mat_vec_kernel_q4g32(half* y, const half *__restrict__ x, PackedSubtensor2DQ4 A) {
  int numCol = A.getNumCol();
  int lda = A.getNumCol();

  int row = blockIdx.x * blockDim.y + threadIdx.y;
  if (row >= A.getNumRow()) return;

  constexpr int VecX = 8;
  constexpr int VecA = 16;
  constexpr int WrapSize = 32;

  int groupPerRow = numCol / Q4::GroupSize;
  constexpr int bytesPerGroup = Q4::GroupSize / 2;
  int rowGroupIdx = row * groupPerRow;
  const uint8_t *__restrict__ pdata = A.getData(row * groupPerRow);

  float sum = 0;
  int groupIdx = threadIdx.x;
  for (int i = groupIdx; i < groupPerRow; i += WrapSize) {
    float scale = float(A.getScaleValue(rowGroupIdx + i));
    float qzero = float(A.getZeroValue(rowGroupIdx + i));

    // 128 elements
    #pragma unroll
    for (int k = 0; k < 1; ++k) {
      uint32_t packA[VecA];
      load16byteCS(&pdata[i * bytesPerGroup], &packA[0]);
      load16byteCS(&pdata[i * bytesPerGroup + 16], &packA[4]);
      load16byteCS(&pdata[i * bytesPerGroup + 32], &packA[8]);
      load16byteCS(&pdata[i * bytesPerGroup + 48], &packA[12]);

      // 32 elements
      #pragma unroll
      for (int j = 0; j < VecA; ++j) {
        uint32_t packAv8 = packA[j];
        half packX[VecX];
        load16byte<half[VecX]>(&x[i * Q4::GroupSize + k * (VecA * VecX) + j * VecX], &packX);

        // 8 elements
        #pragma unroll
        for (int el = 0; el < VecX; ++el) {
          sum += scale * (float(packAv8 & 0xf) - qzero) * float(packX[el]);
          packAv8 = packAv8 >> 4;
        }
      }
    }
  }

  sum = wrapReduceSum(sum);
  if (threadIdx.x == 0)
    y[row] = (half)sum;
}

Tensor gemvHalf(const Tensor &A, const Tensor &B) {
  int n = A.getShape(1);
  int d = A.getShape(0);

  Tensor C = createCudaTensorHalf({d, 1});

  dim3 block_dim(32, 4);
  dim3 grid_dim(divUp(d, 4), 1);

  mat_vec_kernel <<<grid_dim, block_dim, 0 >>> (C.getData<half>(), B.getData<half>(), A.getData<half>(), n, d, n);
  hipDeviceSynchronize();
  return C;
}

Tensor gemvQ4(const Tensor &A, const Tensor &x) {
  CHECK(A.getShape(1) == x.getShape(0) && x.getShape(1) == 1);
  CHECK(x.getShape(0) % Q4::GroupSize == 0);
  int n = A.getShape(1);
  int d = A.getShape(0);

  Tensor C = createCudaTensorHalf({d, 1});

  dim3 block_dim(32, 4);
  dim3 grid_dim(divUp(d, 4), 1);

  mat_vec_kernel_q4g32 <<<grid_dim, block_dim, 0 >>> (C.getData<half>(), x.getData<half>(), A);
  hipDeviceSynchronize();
  return C;
}

}  // cuda
}  // op
}  // ly
